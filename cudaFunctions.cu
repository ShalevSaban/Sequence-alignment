#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "consts.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cFunctions.h"



// implemntaion to atomic change of float (the given operations not valid to float )

__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));


    return old;
}

//helper function to save the best offset , n , k
    __device__ void atomic_change(int *resultArr,int* tempResults)
    {
        atomicExch(&resultArr[BEST_OFFSET],tempResults[BEST_OFFSET]);
        atomicExch(&resultArr[BEST_K_PLACE],tempResults[BEST_M_PLACE]);
        atomicExch(&resultArr[BEST_M_PLACE], tempResults[BEST_K_PLACE]);
    }




//calculate the score of seq1 and given mutant by skip the score of locations n,k
__host__ __device__  float mutantScore(char* seq1,char* seq2 ,int seq2Len,int n,int k,float* scoreMat)
{

    int i=0;
    int j=i;
    float score=0;
    // float matScore=0;
    
    for(i=0;i<seq2Len-2;i++)
    {
        if(j==n || j==k)
            j++;
       
        score+=scoreMat[((seq1[i]-FIRST_CHAR)*NUM_CHARS)+(seq2[j]-FIRST_CHAR)];
    j++;
    }

    return score;

}

//create mutants n,k combitnations by array -for each n the fit k locate at  i + number of mutants (array[i]=n)
__host__ int* createMutants(int num_mutants)
{
	int* nkArr = (int*) malloc(num_mutants * 2 * sizeof(int));

	int n = 0, k = 1;
	for (int i = 0; i < num_mutants; i++) 
	{
		nkArr[i] = n;
		nkArr[i+num_mutants] = k;

		n++;
		if (n == k)
		{
			n = 0;
			k++;
		} 
	}
	
	return nkArr;
}


//calc the best score,mutants ,offsets for seq1,seq2 ,the score stored in pointer and the rest in "result arr"
   __global__ void GPU_bestScore(char* gpu_seq1,char* gpu_seq2,float* gpu_mat,float* score,int* resultArr,int maxOffset,int maxMutant,int seq2Len,int* mutants)
    {
        int i= blockDim.x * blockIdx.x + threadIdx.x;

        int j;
        float currentScore=0;
        float bestScore=-99999;
        int tempResults[RESULT_ARR];

    

        if(i<maxMutant)
        {
            for(j=0;j<maxOffset;j++)
            {
              currentScore=mutantScore(&gpu_seq1[j],&gpu_seq2[0],seq2Len,mutants[i],mutants[i+maxMutant],gpu_mat);

              if(bestScore<currentScore)
              {
                  bestScore=currentScore;
                  tempResults[BEST_OFFSET]=j;
                  tempResults[BEST_M_PLACE]=mutants[i];
                  tempResults[BEST_K_PLACE]=mutants[i+maxMutant];
               }
            }
        //atomic changes to avoid race condition 
        atomic_change(resultArr,tempResults); 
        atomicMaxFloat(score,bestScore);
        }            
    }




 

int computeOnGPU(char* seq1,char* seq2,float* scoreMat,int* resultArr,float* bestScore) {

    int seq2Len=strlen(seq2);
    int seq1Len=strlen(seq1);

    int maxOffset = seq1Len - (seq2Len - 2) + 1;
    int numOfMutants=seq2Len*(seq2Len-1)/2;

    int* mutantsArr=createMutants(numOfMutants);


    //alocate space in gpu 

    char* gpu_seq1;
    MALLOC(gpu_seq1,seq1Len);
    COPY(gpu_seq1,seq1,seq1Len);

    char* gpu_seq2;
    MALLOC(gpu_seq2,seq2Len);
    COPY(gpu_seq2,seq2,seq2Len);

    int matSize=(NUM_CHARS*NUM_CHARS)*sizeof(float);
    float* gpu_mat;
    MALLOC(gpu_mat,matSize);
    COPY(gpu_mat,scoreMat,matSize);

    int* gpu_mutants;
    MALLOC(gpu_mutants,numOfMutants * 2 * sizeof(int));
    COPY(gpu_mutants,mutantsArr,numOfMutants * 2 * sizeof(int));

    float* gpu_score;
    MALLOC(gpu_score,sizeof(float));
    COPY(gpu_score,bestScore,sizeof(float));

    int* result_gpu_arr;
    MALLOC(result_gpu_arr,sizeof(int)*RESULT_ARR);
    COPY(result_gpu_arr,resultArr,sizeof(int)*RESULT_ARR);

 

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numOfMutants + threadsPerBlock - 1) / threadsPerBlock;
    hipError_t err = hipSuccess;
    GPU_bestScore<<<blocksPerGrid, threadsPerBlock>>>(gpu_seq1,gpu_seq2,gpu_mat,gpu_score,result_gpu_arr,maxOffset,numOfMutants,seq2Len,gpu_mutants);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    //copy the results to cpu 
    COPY_HOST(bestScore,gpu_score,sizeof(float));
    COPY_HOST(resultArr,result_gpu_arr,sizeof(int)*RESULT_ARR);


    FREE(gpu_score)
    FREE(result_gpu_arr)
    FREE(gpu_seq1)
    FREE(gpu_seq2)


    return 0;
   
    }

 




 


